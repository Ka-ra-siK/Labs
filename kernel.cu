#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <math.h>
#include <time.h>
#include <locale.h>

#define N 1000
hipError_t err = hipSuccess;
float A[N * (N + 1)];

void check_err(){
	if (err != hipSuccess){
		fprintf(stderr, "Failed ", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


void getMatrix() {
	/*for (int i = 0; i < N; ++i)
		for (int j = 0; j < N; ++j)
			a[i][j] = (float)(rand() % 10);*/
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
			A[i + j * N] = rand() % 10;
		A[i + N * N] = 0;
	}
}

/*__global__ void gaussMethod(float** a) {
	float* temp = (float*)malloc(sizeof(float) * N);
	float buf = 0.0;
	for (int i = 0; i < N - 1; i++)
		for (int j = i + 1; j < N; j++) {
			if (a[i][i] == 0 && i != N - 1) {
				for (int m = 0; m < N; m++) {
					temp[m] = a[i][m];
					a[i][m] = a[i + 1][m];
					a[i + 1][m] = temp[m];
				}
				for (int m = 0; m < N; m++) {
					if (a[i + 1][m] != 0)
						a[i + 1][m] = -a[i + 1][m];
				}
				continue;
			}
			buf = -a[j][i] / a[i][i];
			for (int k = 0; k <= N; k++)
				a[j][k] = a[i][k] * buf + a[j][k];
		}
	free(temp);
	temp = nullptr;
}

__global__ void getDeterminant(float** a) {
	float det = 1.0;
	for (int i = 0; i < N; i++)
		det *= a[i][i];
	//printf("Определитель матрицы = %.3lf\n", det);
}*/

__global__ void gauss_stage1(float* a, int n, float x)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i <= N - n + 1){
		a[n + N * (i + n)] /= x;
	}
}

__global__ void gauss_stage2(float* a, int n, int i, float x){
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if (j <= N - n - 1){
		a[i + N * (j + n + 1)] -= a[n + N * (j + n + 1)] * x;
	}
}

__global__ void gauss_stage3(float* a, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		a[i + N * N] -= a[n + N * N] * a[i + n * N];
	}
}


void findDeterminand() {
	hipError_t err = hipSuccess;

	int threadsPerBlock = 128,
		blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock,
		size = sizeof(float) * N * (N + 1);

	getMatrix();
	float* _A = NULL;
	err = hipMalloc((void**)&_A, size); check_err();
	err = hipMemcpy(_A, A, size, hipMemcpyHostToDevice); check_err();

	for (int i = 0; i < N; i++){
		gauss_stage1 << <blocksPerGrid, threadsPerBlock >> > (_A, i, A[i + i * N]);
		for (int j = i + 1; j < N; j++)
			gauss_stage2 << <blocksPerGrid, threadsPerBlock >> > (_A, i, j, A[j + i * N]);
	}

	for (int i = N - 1; i > 0; i--)
		gauss_stage3 << <blocksPerGrid, threadsPerBlock >> > (_A, i);

	//for (int i = 0; i < N; i++) {
	//	free(A[i]);
	//	A[i] = nullptr;
	//}
	//free(A);
	//A = nullptr;
}

int main(void){
	setlocale(0, "");
	srand(time(NULL));
	int size = N;
	FILE* text;
	int before = clock();
	findDeterminand();
	int time = (clock() - before) * 1000;
	printf("Время: %d мс\n\n", time);

	text = fopen("text.txt", "a");
	fprintf(text, "Время: %d мс     Размер матрицы: %d\n", time, size);
	fclose(text);

	system("pause");
	return  0;
}
