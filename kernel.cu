#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <math.h>
#include <time.h>
#include <locale.h>

#define N 800
hipError_t err = hipSuccess;
float A[N * (N + 1)];

void check_err(){
	if (err != hipSuccess){
		fprintf(stderr, "Failed ", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void getMatrix() {
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
			A[i + j * N] = rand() % 10;
		A[i + N * N] = 0;
	}
}

__global__ void gauss_stage1(float* a, int n, float x){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i <= N - n + 1){
		a[n + N * (i + n)] /= x;
	}
}

__global__ void gauss_stage2(float* a, int n, int i, float x){
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if (j <= N - n - 1){
		a[i + N * (j + n + 1)] -= a[n + N * (j + n + 1)] * x;
	}
}

__global__ void gauss_stage3(float* a, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n){
		a[i + N * N] -= a[n + N * N] * a[i + n * N];
	}
}


void findDeterminand() {
	hipError_t err = hipSuccess;

	int threadsPerBlock = 128,
		blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock,
		size = sizeof(float) * N * (N + 1);

	getMatrix();
	float* _A = NULL;
	err = hipMalloc((void**)&_A, size); check_err();
	err = hipMemcpy(_A, A, size, hipMemcpyHostToDevice); check_err();

	for (int i = 0; i < N; i++){
		gauss_stage1 << <blocksPerGrid, threadsPerBlock >> > (_A, i, A[i + i * N]);
		for (int j = i + 1; j < N; j++)
			gauss_stage2 << <blocksPerGrid, threadsPerBlock >> > (_A, i, j, A[j + i * N]);
	}

	for (int i = N - 1; i > 0; i--)
		gauss_stage3 << <blocksPerGrid, threadsPerBlock >> > (_A, i);

	hipFree(_A);
}

int main(void){
	setlocale(0, "");
	srand(time(NULL));
	int size = N;
	FILE* text;
	int before = clock();
	findDeterminand();
	int time = (clock() - before) * 1000;
	printf("Время: %d мс\n\n", time);

	text = fopen("text.txt", "a");
	fprintf(text, "Размер матрицы: %d     Время: %d мс\n", size, time);
	fclose(text);

	system("pause");
	return  0;
}
